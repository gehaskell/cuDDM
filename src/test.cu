#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include <opencv2/opencv.hpp>
using namespace cv;

// Test file to see where we are going wrong

__global__ void Test_AbsDifference(hipfftReal *d_diff, float *d_frame1, float *d_frame2, int width, int height) {
	int x = threadIdx.x + blockIdx.x * 32;
	int y = threadIdx.y + blockIdx.y * 32;

	if (x <= width-1 && y <= height-1) {
		int pos_offset = y * width + x;
		d_diff[pos_offset] = abs(d_frame1[pos_offset] - d_frame2[pos_offset]);
	}
	return;
}

__global__ void Test_processFFT(hipfftComplex *d_data, float *d_fft, int width, int height) {
	// Takes output of cuFFT R2C operation, normalises it (i.e. divides by px count), takes the magnitude and adds it to the accum_array

	int size = width * height;

	int j = threadIdx.x + blockIdx.x * 32;
	int i = threadIdx.y + blockIdx.y * 32;

	float mag;
	if (j <= width-1 && i <= height-1) {
		int pos_offset = i * width + j;
		int sym_w = width / 2 + 1; // to deal with complex (hermitian) symmetry

		if (j >= sym_w) {
			// real ->  d_data[i*sym_w+(width-j)].x
			// img  -> -d_data[i*sym_w+(width-j)].y
			mag = hipCabsf(d_data[i*sym_w+(width-j)]);

		} else {
			// real -> d_data[i*sym_w+j].x
			// img  -> d_data[i*sym_w+j].y
			mag = hipCabsf(d_data[i*sym_w+j]);
		}

		// add to fft_accum
		d_fft[pos_offset] = mag;
	}
}


int _main() {
	VideoCapture cap("/home/ghaskell/projects_Git/cuDDM/data/colloid_0.5um_vid.mp4");

	Mat img1;
	Mat img2;

	int w = 5;
	int h = 5;
	int delta = 3;

	float* h_frame1 = new float[w * h];
	float* h_frame2 = new float[w * h];

	cap >> img1;
	while (delta >= 0) {
		cap >> img2;
		delta--;
	}

	for (int y = 0; y < h; y++) {
		for (int x = 0; x < w; x++) {
			h_frame1[y * w + x] = (float) img1.data[((img1.step)/img1.elemSize1())* y + img1.channels() * x];
			h_frame2[y * w + x] = (float) img2.data[((img2.step)/img2.elemSize1())* y + img2.channels() * x];
		}
	}

	float *d_frame1, *d_frame2;
	hipMalloc((void **) &d_frame1, w * h * sizeof(float));
	hipMalloc((void **) &d_frame2, w * h * sizeof(float));
	hipMemcpy(d_frame1, h_frame1,  w * h * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_frame2, h_frame2,  w * h * sizeof(float), hipMemcpyHostToDevice);

	hipfftReal *d_diff_local;
	hipMalloc((void **) &d_diff_local, w * h * sizeof(hipfftReal));
	hipfftReal *h_diff_local = new hipfftReal[w * h];

	dim3 blockDim(32, 32, 1);
	dim3 gridDim((int)ceil(w/32.0), (int)ceil(h/32.0), 1);

	Test_AbsDifference<<<gridDim, blockDim>>>(d_diff_local, d_frame1, d_frame2, w, h);
	hipMemcpy(h_diff_local, d_diff_local,  w * h * sizeof(hipfftReal), hipMemcpyDeviceToHost);

	// FFT
	hipfftHandle plan;
	if ((hipfftPlan2d(&plan, w, h, HIPFFT_R2C)) != HIPFFT_SUCCESS) {
		std::cout << "cufft plan error" << std::endl;
	}
	hipfftComplex *d_fft_local; // should i malloc this????
	hipMalloc((void **) &d_fft_local, w * (h/2 + 1) * sizeof(hipfftComplex));
	if ((hipfftExecR2C(plan, d_diff_local, d_fft_local)) != HIPFFT_SUCCESS) {
		std::cout << "cufft exec error" << std::endl;
	}

	float *d_fft;
	hipMalloc((void **) &d_fft, w * h * sizeof(float));
	float *h_fft = new float[w*h];
	Test_processFFT<<<gridDim, blockDim>>>(d_fft_local, d_fft, w, h);
	hipMemcpy(h_fft, d_fft,  w * h * sizeof(float), hipMemcpyDeviceToHost);


	for (int y = 0; y < h; y++) {
		for (int x = 0; x < w; x++) {
			std::cout << h_frame1[y*w + x] << ","<< h_frame2[y*w + x] << ","<< h_diff_local[y*w + x] << "," << h_fft[y*w + x]<< " " << std::endl;
		}
	}





	std::cout << "END" << std::endl;


}
