#include "hip/hip_runtime.h"
//TODO: clean up print statements - switch errors to fprintf
//TODO: probably causes memory leak

#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <fstream>
#include <opencv2/opencv.hpp>
#include <chrono>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

using namespace cv;

__global__ void AbsDifference(hipfftReal *d_diff, float *d_frame1, float *d_frame2, int width, int height) {
	int x = threadIdx.x + blockIdx.x * 32;
	int y = threadIdx.y + blockIdx.y * 32;

	if (x <= width-1 && y <= height-1) {
		int pos_offset = y * width + x;
		d_diff[pos_offset] = abs(d_frame1[pos_offset] - d_frame2[pos_offset]);
	}
	return;
}


__global__ void processFFT(hipfftComplex *d_data, float *d_fft, int tau_idx, int width, int height) {
	// Takes output of cuFFT R2C operation, normalises it (i.e. divides by px count), takes the magnitude and adds it to the accum_array

	int size = width * height;

	int j = threadIdx.x + blockIdx.x * 32;
	int i = threadIdx.y + blockIdx.y * 32;

	float mag;
	if (j <= width-1 && i <= height-1) {
		int pos_offset = i * width + j;
		int sym_w = width / 2 + 1; // to deal with complex (hermitian) symmetry

		if (j >= sym_w) {
			// real ->  d_data[i*sym_w+(width-j)].x
			// img  -> -d_data[i*sym_w+(width-j)].y
			mag = hipCabsf(d_data[i*sym_w+(width-j)]) / (float)size;

		} else {
			// real -> d_data[i*sym_w+j].x
			// img  -> d_data[i*sym_w+j].y
			mag = hipCabsf(d_data[i*sym_w+j]) / (float)size;
		}

		// add to fft_accum
		d_fft[tau_idx * size + pos_offset] += mag*mag;
	}
}


bool LoadVideoToBuffer(float *h_ptr, int frame_count, VideoCapture cap, int w, int h) {
	//printf("load video (%d frames) (w: %d, h: %d)\n", frame_count, w, h);

	// No bounds check! assume that w, h smaller than mat
	int num_elements = w * h;

	Mat input_img; //, grayscale_img;

	// There is some problems with the image type we are using - though some effort was put into switching to a
	// more generic image format, more thought is required therefore switch to just dealing with 3 channel uchars
	// look at http://ninghang.blogspot.com/2012/11/list-of-mat-type-in-opencv.html and
	// https://docs.opencv.org/3.4/d3/d63/classcv_1_1Mat.html#aa5d20fc86d41d59e4d71ae93daee9726 for more info.


	for (int frame_idx = 0; frame_idx < frame_count; frame_idx++) {
		//std::cout << "Loaded frame " << frame_idx << std::endl;

		cap >> input_img;

		if (input_img.empty()) {
			fprintf(stderr,"Video frame is empty");
			return false;
		}

		//input_img.convertTo(grayscale_img, CV_32FC1); // covert to grayscale image

		if (input_img.type() != 16) {
			std::cout << "Non standard image format detected, may cause unexpected behaviour, image type : " << input_img.type() << std::endl;
			return false;
		}

	    //imshow("Input", input_img);
	    //waitKey(0);

		int cols = input_img.cols, rows = input_img.rows;

		if (w > cols || h > rows) {
			std::cout << "Issue: specified width / height > cols / rows." << std::endl;
		}

		for (int y = 0; y < h; y++) {
			for (int x = 0; x < w; x++) {
				// Using img.at<>8
				h_ptr[frame_idx * num_elements + y * w + x] =  (float) input_img.data[((input_img.step)/input_img.elemSize1())* y + input_img.channels() * x];
			}
		}
	}
	return true;
}


void processChunk(hipStream_t stream, float *d_ptr, int frame_count, float *d_out, int *tau_vector, int tau_count, int width, int height, int repeat_count = 20, float *debug_buff=NULL) {
	// debug_buffer is a width * height *sizeof(float) buffer which can be printed
	//	if (debug_buff != NULL) {
	//		hipMemcpy(debug_buff, <device ptr>, width*height*sizeof(float), hipMemcpyDeviceToHost);
	//		return;
	//	}
	// d_out size: tau_count * width * height * sizeof(float)

	int w = width;
	int h = height;

	//printf("chunk analysis (%d frames).\n", frame_count);

	// Max 1024 (32 x 32) threads per block hence multiple blocks to operate on a frame
	// Max number of thread blocks is 65536)

	dim3 blockDim(32, 32, 1);
	dim3 gridDim((int) ceil(width/32.0), (int) ceil(height/32.0), 1);

	if ((int) ceil(width/32.0) * (int) ceil(height/32.0) > 65536) {
		fprintf(stderr, "Image too big, not enough thread blocks (%d).\n", (int) ceil(width/32.0) * (int) ceil(height/32.0));
	}


	hipfftReal *d_local_absdiff;
	hipMalloc((void **) &d_local_absdiff, w * h * sizeof(hipfftReal));

	hipfftComplex *d_local_fft;
	hipMalloc((void **) &d_local_fft, w * (h / 2 + 1) * sizeof(hipfftComplex));

	// cuFFT plan
	hipfftHandle plan;
	if ((hipfftPlan2d(&plan, w, h, HIPFFT_R2C)) != HIPFFT_SUCCESS) {
		fprintf(stderr, "cuFFT Error: Plan failure.\n");
	}
	hipfftSetStream(plan, stream);


	// Main loop
	int tau, idx1, idx2;
	float *d_frame1, *d_frame2;

	for (int repeat = 0; repeat < repeat_count; repeat++) {
		for (int tau_idx = 0; tau_idx < tau_count; tau_idx++) {
			tau = tau_vector[tau_idx];

			idx1 = rand() % (frame_count - tau);
			idx2 = idx1 + tau;
			//std::cout << "tau: " << tau << " idxs: " << idx1 << ", " << idx2 << std::endl;

			d_frame1 = d_ptr + (idx1 * w * h);	// float pointer to frame 1
			d_frame2 = d_ptr + (idx2 * w * h);

			AbsDifference<<<gridDim, blockDim, 0, stream >>>(d_local_absdiff, d_frame1, d_frame2, w, h); // find absolute difference

			//FFT execute
			if ((hipfftExecR2C(plan, d_local_absdiff, d_local_fft)) != HIPFFT_SUCCESS) {
				std::cout << "cuFFT Exec Error\n" << std::endl;
			}

			processFFT<<<gridDim, blockDim, 0, stream>>>(d_local_fft, d_out, tau_idx, w, h); // process FFT (i.e. normalise and add to accumulator)
		}
	}
	hipFree(d_local_absdiff); hipFree(d_local_fft);
	hipfftDestroy(plan);

	return;
}


int main(){
	for (int x = 0; x < 5; x++) {
		auto t1 = std::chrono::high_resolution_clock::now();

		int w = 1024;
		int h = 1024;
		int total_frames = 500;
		int buffer_frames = 50;
		int tau_count = 15;
		int tau_vector [tau_count] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15};
		VideoCapture cap("/home/ghaskell/projects_Git/cuDDM/data/test.mp4");

		// Initialisation
		int iterations = total_frames / buffer_frames;
		bool read_ok;

		float *h_buffer1, *h_buffer2;
		float *d_buffer1, *d_buffer2;
		float *d_out;

		int buffer_size = sizeof(float) * buffer_frames * w * h;

		gpuErrchk(hipHostAlloc((void **) &h_buffer1, buffer_size, hipHostMallocDefault));
		gpuErrchk(hipHostAlloc((void **) &h_buffer2, buffer_size, hipHostMallocDefault));
		gpuErrchk(hipMalloc((void **) &d_buffer1, buffer_size));
		gpuErrchk(hipMalloc((void **) &d_buffer2, buffer_size));
		gpuErrchk(hipMalloc((void **) &d_out, sizeof(float) * tau_count* w * h));

		hipStream_t stream1, stream2;
		hipStreamCreate(&stream1); hipStreamCreate(&stream2);

		float *d_data = d_buffer1;
		float *h_data = h_buffer1;

		float *d_next = d_buffer2;
		float *h_next = h_buffer2;

		hipStream_t *work_stream = &stream1;
		hipStream_t *next_stream = &stream2;

		read_ok = LoadVideoToBuffer(h_data, buffer_frames, cap, w, h); // puts chunk data into pinned host memory

		while (read_ok && iterations > 0) {
			gpuErrchk(hipMemcpyAsync(d_data, h_data, buffer_size, hipMemcpyHostToDevice, *work_stream)); // copy buffer to device

			// PROCESS FRAME - use work stream
			processChunk(*work_stream, d_data, buffer_frames, d_out, tau_vector, tau_count, w, h);

			gpuErrchk(hipStreamSynchronize(*next_stream)); // prevent overrun

			read_ok = LoadVideoToBuffer(h_next, buffer_frames, cap, w, h);

			// Swap working and secondary streams
			float *tmp = h_data;
			h_data = h_next;
			h_next = tmp;

			tmp = d_data;
			d_data = d_next;
			d_next = tmp;

			hipStream_t *st_tmp = work_stream;
			work_stream = next_stream;
			next_stream = st_tmp;

			printf("Interation complete (Iterations = %d))\n", iterations);
			iterations--;

		}
		printf("Done\n");

		auto t2 = std::chrono::high_resolution_clock::now();
		auto duration = std::chrono::duration_cast<std::chrono::microseconds>( t2 - t1 ).count();
		std::cout << (float)duration/1000000.0 << std::endl;

	}
}
