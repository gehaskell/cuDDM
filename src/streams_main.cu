#include "hip/hip_runtime.h"
//TODO: clean up print statements - switch errors to fprintf
//TODO: probably causes memory leak

#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <fstream>
#include <opencv2/opencv.hpp>
#include <chrono>
#include <nvToolsExt.h>

#define blockSize_x 16
#define blockSize_y 16

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

using namespace cv;

__global__ void AbsDifference(hipfftReal *d_diff, float *d_frame1, float *d_frame2, int width, int height) {
	int x = threadIdx.x + blockIdx.x * blockSize_x;
	int y = threadIdx.y + blockIdx.y * blockSize_y;

	if (x <= width-1 && y <= height-1) {
		int pos_offset = y * width + x;
		d_diff[pos_offset] = abs(d_frame1[pos_offset] - d_frame2[pos_offset]);
	}
	return;
}


__global__ void processFFT(hipfftComplex *d_data, float *d_fft, int tau_idx, int width, int height) {
	// Takes output of cuFFT R2C operation, normalises it (i.e. divides by px count), takes the magnitude and adds it to the accum_array

	int size = width * height;

	int j = threadIdx.x + blockIdx.x * blockSize_x;
	int i = threadIdx.y + blockIdx.y * blockSize_y;

	float mag;
	if (j <= width-1 && i <= height-1) {
		int pos_offset = i * width + j;
		int sym_w = width / 2 + 1; // to deal with complex (hermitian) symmetry

		if (j >= sym_w) {
			// real ->  d_data[i*sym_w+(width-j)].x
			// img  -> -d_data[i*sym_w+(width-j)].y
			mag = hipCabsf(d_data[i*sym_w+(width-j)]) / (float)size;

		} else {
			// real -> d_data[i*sym_w+j].x
			// img  -> d_data[i*sym_w+j].y
			mag = hipCabsf(d_data[i*sym_w+j]) / (float)size;
		}

		// add to fft_accum
		d_fft[tau_idx * size + pos_offset] += mag*mag;
	}
}


bool LoadVideoToBuffer(float *h_ptr, int frame_count, VideoCapture cap, int w, int h) {
	nvtxRangePush(__FUNCTION__); // to track video loading times in nvvp

	//printf("load video (%d frames) (w: %d, h: %d)\n", frame_count, w, h);

	// No bounds check! assume that w, h smaller than mat
	int num_elements = w * h;

	Mat input_img; //, grayscale_img;

	// There is some problems with the image type we are using - though some effort was put into switching to a
	// more generic image format, more thought is required therefore switch to just dealing with 3 channel uchars
	// look at http://ninghang.blogspot.com/2012/11/list-of-mat-type-in-opencv.html and
	// https://docs.opencv.org/3.4/d3/d63/classcv_1_1Mat.html#aa5d20fc86d41d59e4d71ae93daee9726 for more info.


	for (int frame_idx = 0; frame_idx < frame_count; frame_idx++) {
		//std::cout << "Loaded frame " << frame_idx << std::endl;

		cap >> input_img;

		if (input_img.empty()) {
			fprintf(stderr,"Video frame is empty");
			return false;
		}

		//input_img.convertTo(grayscale_img, CV_32FC1); // covert to grayscale image

		if (input_img.type() != 16) {
			std::cout << "Non standard image format detected, may cause unexpected behaviour, image type : " << input_img.type() << std::endl;
			return false;
		}

	    //imshow("Input", input_img);
	    //waitKey(0);

		int cols = input_img.cols, rows = input_img.rows;

		if (w > cols || h > rows) {
			std::cout << "Issue: specified width / height > cols / rows." << std::endl;
		}

		for (int y = 0; y < h; y++) {
			for (int x = 0; x < w; x++) {
				// Using img.at<>8
				h_ptr[frame_idx * num_elements + y * w + x] =  (float) input_img.data[((input_img.step)/input_img.elemSize1())* y + input_img.channels() * x];
			}
		}
	}

    nvtxRangePop();
	return true;
}


void processChunk(hipStream_t stream, float *d_ptr,
		int frame_count,
		float *d_out,
		int *tau_vector,
		int tau_count,
		hipfftReal *d_abs_workspace,
		hipfftComplex *d_fft_workspace,
		hipfftHandle plan,
		int width, int height,
		int repeat_count = 50, float *debug_buff=NULL) {

	// debug_buffer is a width * height *sizeof(float) buffer which can be printed
	//	if (debug_buff != NULL) {
	//		hipMemcpy(debug_buff, <device ptr>, width*height*sizeof(float), hipMemcpyDeviceToHost);
	//		return;
	//	}
	// d_out size: tau_count * width * height * sizeof(float)

	int w = width;
	int h = height;

	//printf("chunk analysis (%d frames).\n", frame_count);

	// Max 1024 (32 x 32) threads per block hence multiple blocks to operate on a frame
	// Max number of thread blocks is 65536)

	dim3 blockDim(blockSize_x, blockSize_y, 1);
	int grid_x = (int) ceil(width/(float)blockSize_x);
	int grid_y = (int) ceil(width/(float)blockSize_y);

	dim3 gridDim(grid_x, grid_y, 1);

	if (gridDim.x * gridDim.y * gridDim.z > 65536) {
		fprintf(stderr, "Image too big, not enough thread blocks (%d).\n", gridDim.x * gridDim.y * gridDim.z);
	}

	hipfftSetStream(plan, stream);

	// Main loop
	int tau, idx1, idx2;
	float *d_frame1, *d_frame2;
	hipfftComplex *d_local_fft;
	hipfftReal *d_local_absdiff;

	for (int repeat = 0; repeat < repeat_count; repeat++) {
		for (int tau_idx = 0; tau_idx < tau_count; tau_idx++) {
			tau = tau_vector[tau_idx];

			d_local_fft = d_fft_workspace + (tau_idx * w * (h / 2 + 1));
			d_local_absdiff = d_abs_workspace + (tau_idx * w * h);

			idx1 = rand() % (frame_count - tau);
			idx2 = idx1 + tau;
			//std::cout << "tau: " << tau << " idxs: " << idx1 << ", " << idx2 << std::endl;

			d_frame1 = d_ptr + (idx1 * w * h);	// float pointer to frame 1
			d_frame2 = d_ptr + (idx2 * w * h);

			AbsDifference<<<gridDim, blockDim, 0, stream >>>(d_local_absdiff, d_frame1, d_frame2, w, h); // find absolute difference

			//FFT execute
			if ((hipfftExecR2C(plan, d_local_absdiff, d_local_fft)) != HIPFFT_SUCCESS) {
				std::cout << "cuFFT Exec Error\n" << std::endl;
			}

			processFFT<<<gridDim, blockDim, 0, stream>>>(d_local_fft, d_out, tau_idx, w, h); // process FFT (i.e. normalise and add to accumulator)
		}
	}

	return;
}


void HARDCODEanalyseFFTHost(float *d_in, int norm_factor, int *tau_vector, int tau_count, int width, int height) {
    int w = width; int h = height;

	// Generate q - vectors - Hard Coded
	int q_count = 50;

	float q_squared[q_count];
	float q_vector[q_count];

	for (int i = 0; i < q_count; i++) {
		//std::cout << 50 * ((float)i /20.0) << std::endl;
		q_vector[i] = 100 * ((float)(i+1) /20.0);
		q_squared[i] = q_vector[i] * q_vector[i];
	}

	// Generate masks
    int *px_count = new int[q_count](); // () initialises to zero
    float *masks = new float[w * h * q_count];

    float half_w, half_h;
    half_h = height / 2.0;
    half_w = width / 2.0;
    float r_sqr, ratio;

    // First Generate the radius masks
    int shift_x, shift_y;
    for (int q_idx = 0; q_idx < q_count; q_idx++) {
        for (int x = 0; x < w; x++)
        {
            for (int y = 0; y < h; y++)
            {
                // Perform manual FFT shift
                shift_x = (x + (int)half_w) % w;
                shift_y = (y + (int)half_h) % h;

                // Distance relative to centre
                shift_x -= half_w;
                shift_y -= half_h;

                r_sqr = shift_x * shift_x + shift_y * shift_y;
                ratio = r_sqr / q_squared[q_idx];

                if (1 <= ratio && ratio <= 1.44) { // we want values from 1.0 * q to 1.2 * q
                    masks[q_idx*w*h + y*w + x] = 1.0;
                    px_count[q_idx] += 1;
                } else {
                    masks[q_idx*w*h + y*w + x] = 0.0;
                }
            }
        }
    }


    // Start analysis
    float val;
	float * iq_tau = new float[tau_count * q_count]();

    for (int tau_idx = 0; tau_idx < tau_count; tau_idx++) {

        for (int q_idx = 0; q_idx < q_count; q_idx++) {
        	val = 0;
        	int px = 0;

        	if (px_count[q_idx] != 0) { // If the mask has no values iq_tau must be zero

        		for (int i = 0; i < w*h; i++) { 	// iterate through all pixels
                	val += d_in[w * h * tau_idx + i] * masks[w * h * q_idx + i];
                	if (masks[w * h * q_idx + i]) {px += 1;}
                }
                // Also should divide by chunk count
                val /= (float)px_count[q_idx]; // could be potential for overflow here
                val /= (float)norm_factor;
        	}

        	iq_tau[q_idx * tau_count + tau_idx] = val;
        }
    }

	// outputting iqtau
    std::ofstream myfile("/home/ghaskell/projects_Git/cuDDM_streams/data/iqt.txt");

    if (myfile.is_open()) {
    	for (int i = 0; i < q_count; i++) {
    		myfile << q_vector[i] << " ";
    	}
		myfile << "\n";
    	for (int i = 0; i < tau_count; i++) {
    		myfile << tau_vector[i] << " ";
    	}
		myfile << "\n";

		for (int q_idx = 0; q_idx < q_count; q_idx++) {
	    	for (int t_idx = 0; t_idx < tau_count; t_idx++) {
	    		myfile << iq_tau[q_idx * tau_count + t_idx] << " ";
	    	}
			myfile << "\n";
		}

		myfile.close();
    } else {
    	std::cout << "Unable to open file" << std::endl;
    	return;
    }
}



int main(){
	for (int x = 0; x < 1; x++) {
		printf("Grid dimensions %d, %d, %d\n", blockSize_x, blockSize_y, 1);

		auto t1 = std::chrono::high_resolution_clock::now();

		int w = 1024;
		int h = 1024;
		int buffer_frames = 20;
		int total_frames = 200;
		int tau_count = 15;
		int tau_vector [tau_count] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15};
		int repeat_count = 50;
		VideoCapture cap("/home/ghaskell/projects_Git/cuDDM/data/test.mp4");

		// Initialisation
		int iterations = total_frames / buffer_frames;
		int kiterations = iterations;
		bool read_ok;

		float *h_buffer1, *h_buffer2;
		float *d_buffer1, *d_buffer2;
		float *h_out, *d_out;

		int buffer_size = sizeof(float) * buffer_frames * w * h;

		gpuErrchk(hipHostAlloc((void **) &h_buffer1, buffer_size, hipHostMallocDefault));
		gpuErrchk(hipHostAlloc((void **) &h_buffer2, buffer_size, hipHostMallocDefault));

		gpuErrchk(hipMalloc((void **) &d_buffer1, buffer_size));
		gpuErrchk(hipMalloc((void **) &d_buffer2, buffer_size));

		gpuErrchk(hipMalloc((void **) &d_out, sizeof(float) * tau_count* w * h));
		h_out = new float[tau_count* w * h];

		float *d_data = d_buffer1;
		float *h_data = h_buffer1;

		float *d_next = d_buffer2;
		float *h_next = h_buffer2;

		hipStream_t stream1, stream2;
		hipStreamCreate(&stream1); hipStreamCreate(&stream2);
		hipStream_t *work_stream = &stream1;
		hipStream_t *next_stream = &stream2;

		// Workspace
		hipfftReal *d_abs_workspace1;
		hipfftReal *d_abs_workspace2;
		hipMalloc((void **) &d_abs_workspace1, tau_count * w * h * sizeof(hipfftReal));
		hipMalloc((void **) &d_abs_workspace2, tau_count * w * h * sizeof(hipfftReal));

		hipfftComplex *d_fft_workspace1;
		hipfftComplex *d_fft_workspace2;
		hipMalloc((void **) &d_fft_workspace1, tau_count * w * (h / 2 + 1) * sizeof(hipfftComplex));
		hipMalloc((void **) &d_fft_workspace2, tau_count * w * (h / 2 + 1) * sizeof(hipfftComplex));

		hipfftComplex *d_fft_current = d_fft_workspace1;
		hipfftComplex *d_fft_next = d_fft_workspace2;

		hipfftReal *d_abs_current = d_abs_workspace1;
		hipfftReal *d_abs_next = d_abs_workspace1;

		// cuFFT plan
		hipfftHandle plan;
		if ((hipfftPlan2d(&plan, w, h, HIPFFT_R2C)) != HIPFFT_SUCCESS) {
			fprintf(stderr, "cuFFT Error: Plan failure.\n");
		}

		// Main loop

		read_ok = LoadVideoToBuffer(h_data, buffer_frames, cap, w, h); // puts chunk data into pinned host memory

		while (read_ok && iterations > 0) {
			gpuErrchk(hipMemcpyAsync(d_data, h_data, buffer_size, hipMemcpyHostToDevice, *work_stream)); // copy buffer to device

			// PROCESS FRAME - use work stream
			processChunk(*work_stream, d_data, buffer_frames, d_out, tau_vector, tau_count, d_abs_current, d_fft_current, plan, w, h, repeat_count); // repeat count optional

			gpuErrchk(hipStreamSynchronize(*next_stream)); // prevent overrun

			read_ok = LoadVideoToBuffer(h_next, buffer_frames, cap, w, h); // load next while GPU processing current

			// Swap working and secondary streams
			float *tmp = h_data;
			h_data = h_next;
			h_next = tmp;

			tmp = d_data;
			d_data = d_next;
			d_next = tmp;

			hipStream_t *st_tmp = work_stream;
			work_stream = next_stream;
			next_stream = st_tmp;

			hipfftComplex *fft_tmp = d_fft_current;
			d_fft_current = d_fft_next;
			d_fft_next = fft_tmp;

			hipfftReal *abs_tmp = d_abs_current;
			d_abs_current = d_abs_next;
			d_abs_next = abs_tmp;

			//printf("chunk complete (%d \\ %d))\n", kiterations- iterations + 1, kiterations);
			iterations--;

		}

		gpuErrchk(hipMemcpy(h_out, d_out, sizeof(float) * tau_count* w * h, hipMemcpyDeviceToHost));

		hipFree(h_buffer1); hipFree(h_buffer2);
		hipFree(d_buffer1); hipFree(d_buffer2);
		hipFree(d_out);
		hipFree(d_abs_workspace1); hipFree(d_abs_workspace2);
		hipFree(d_fft_workspace1); hipFree(d_fft_workspace2);
		hipfftDestroy(plan);


		//printf("Done\n");

		auto t2 = std::chrono::high_resolution_clock::now();
		auto duration = std::chrono::duration_cast<std::chrono::microseconds>( t2 - t1 ).count();
		std::cout << "END (time elapsed: " << (float)duration/1000000.0 << " seconds.)"<< std::endl;

		HARDCODEanalyseFFTHost(h_out, repeat_count*kiterations, tau_vector, tau_count, w, h);



	}
}
