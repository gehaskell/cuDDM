#include <hip/hip_runtime.h>

void AnalyseBuffer()

void RunDDM() {
	//
	chunks_per_buffer = 5;
	buffer_frame_length = 30;

	// Initialise workspace
	int w, int h, int frame_count;
	frame_count = chunks_per_buffer * buffer_frame_length;

	int mem_size = sizeof(float) * w * h * frame_count;

	float *d_data1, *d_data2;
	checkCudaErrors(hipMalloc((void**) &d_data1, mem_size));
	checkCudaErrors(hipMalloc((void**) &d_data2, mem_size));

	hipStream_t stream1, stream2;
	hipStreamCreate(stream1);
	hipStreamCreate(stream2);

	hipEvent_t data1_read, data2_read;

	LoadBuffer(stream1, d_data1);
	for (;;) {
		Analyse(stream1, d_data1);
		LoadBuffer(stream2, d_data2);

		//hipStreamWaitEvent(stream,event)
		//hipStreamWaitEvent(stream,event)

		LoadBuffer(stream1, d_data1);
		Analyse(stream2, d_data2);

		//hipStreamWaitEvent(stream,event)
		//hipStreamWaitEvent(stream,event)
	}
	Analyse(stream1, d_data1);
}

int main(int argc, char **argv) {

}
