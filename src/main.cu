#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <opencv2/opencv.hpp>
#include <fstream>

using namespace cv;

struct VideoInfo {
    int width;
    int height;
    int frame_count;
};

struct QMaskStruct {
    // encapsulates data surrounding q-vector mask

    int q_count; // count of q vectors
    float * mask_fft; // FFT of q vector mask
    float * q_vector; // average q associated with each mask
    int * px_count; // normalisation factor for each mask
};

void write_to_file(float * ptr, int width, int height) {
	std::ofstream myfile("/home/ghaskell/projects_Git/cuDDM/data/data2.txt");
	float * print_buff = new float[width * height];
	if (myfile.is_open()) {
		for (int x = 0; x < width*height; x++) {
			myfile << ptr[x] <<" ";
		}
		myfile << std::endl;
	}
	myfile.close();
}


void LoadVideoToBuffer(float *d_ptr, int frame_count, VideoCapture cap, int w, int h) {
	std::cout << "Load frame " << frame_count << " (w: " <<  w << " h: " << h << ")" << std::endl;

	// No bounds check! assume that w, h smaller than mat
	int num_elements = w * h;

	Mat input_img; //, grayscale_img;
	float *h_ptr = new float[w * h * frame_count];

	// There is some problems with the image type we are using - though some effort was put into switching to a
	// more generic image format, more thought is required therefore switch to just dealing with 3 channel uchars
	// look at http://ninghang.blogspot.com/2012/11/list-of-mat-type-in-opencv.html and
	// https://docs.opencv.org/3.4/d3/d63/classcv_1_1Mat.html#aa5d20fc86d41d59e4d71ae93daee9726 for more info.


	for (int frame_idx = 0; frame_idx < frame_count; frame_idx++) {
		//std::cout << "Loaded frame " << frame_idx << std::endl;

		cap >> input_img;

		if (input_img.empty()) {
			std::cout << "Loaded frame is empty." << std::endl;
		}

		//input_img.convertTo(grayscale_img, CV_32FC1); // covert to grayscale image

		if (input_img.type() != 16) {
			std::cout << "Non standard image format detected, may cause unexpected behaviour, image type : " << input_img.type() << std::endl;
		}

	    //imshow("Input", input_img);
	    //waitKey(0);

		int cols = input_img.cols, rows = input_img.rows;

		if (w > cols || h > rows) {
			std::cout << "Issue: specified width / height > cols / rows." << std::endl;
		}

		for (int y = 0; y < h; y++) {
			for (int x = 0; x < w; x++) {
				h_ptr[frame_idx * num_elements + y * w + x] =  (float) input_img.data[((input_img.step)/input_img.elemSize1())* y + input_img.channels() * x];
			}
		}
	}
	hipMemcpy(d_ptr, h_ptr, num_elements * frame_count * sizeof(float), hipMemcpyHostToDevice);
}

//__global__ void AbsDifference(float *d_buffer, hipfftReal *d_diff, int frame1, int frame2, int width, int height) {
//	int size = width * height;
//
//	int x = threadIdx.x + blockIdx.x * 32;
//	int y = threadIdx.y + blockIdx.y * 32;
//
//	if (x <= width-1 && y <= height-1) {
//		int pos_offset = y * width + x;
//		d_diff[pos_offset] = abs(d_buffer[frame1 * size + pos_offset] - d_buffer[frame2 * size + pos_offset]);
//	}
//
//	return;
//}

__global__ void AbsDifference(hipfftReal *d_diff, float *d_frame1, float *d_frame2, int width, int height) {
	int x = threadIdx.x + blockIdx.x * 32;
	int y = threadIdx.y + blockIdx.y * 32;

	if (x <= width-1 && y <= height-1) {
		int pos_offset = y * width + x;
		d_diff[pos_offset] = abs(d_frame1[pos_offset] - d_frame2[pos_offset]);
	}
	return;
}


__global__ void processFFT(hipfftComplex *d_data, float *d_fft, int tau_idx, int width, int height) {
	// Takes output of cuFFT R2C operation, normalises it (i.e. divides by px count), takes the magnitude and adds it to the accum_array

	int size = width * height;

	int j = threadIdx.x + blockIdx.x * 32;
	int i = threadIdx.y + blockIdx.y * 32;

	float mag;
	if (j <= width-1 && i <= height-1) {
		int pos_offset = i * width + j;
		int sym_w = width / 2 + 1; // to deal with complex (hermitian) symmetry

		if (j >= sym_w) {
			// real ->  d_data[i*sym_w+(width-j)].x
			// img  -> -d_data[i*sym_w+(width-j)].y
			mag = hipCabsf(d_data[i*sym_w+(width-j)]) / (float)size;

		} else {
			// real -> d_data[i*sym_w+j].x
			// img  -> d_data[i*sym_w+j].y
			mag = hipCabsf(d_data[i*sym_w+j]) / (float)size;
		}

		// add to fft_accum
		d_fft[tau_idx * size + pos_offset] += mag*mag;
	}
}

void analyseChunk(float *d_chunk_ptr, int frame_count, float *d_out, int *tau_vector, int tau_count, int width, int height, float *debug_buff=NULL) {
	// debug_buffer is a width * height *sizeof(float) buffer which can be printed
	//	if (debug_buff != NULL) {
	//		hipMemcpy(debug_buff, <device ptr>, width*height*sizeof(float), hipMemcpyDeviceToHost);
	//		return;
	//	}
	// d_out size: tau_count * width * height * sizeof(float)

	int w = width;
	int h = height;

	std::cout << "Chunk Analysis Start (" << frame_count << " frames)" <<  std::endl;

	// Initialise workspace
	// these buffers are fit for one frame, if we do all taus at once then should modify

	hipfftReal *d_diff_local;
	hipMalloc((void **) &d_diff_local, w * h * sizeof(hipfftReal));

	hipfftComplex *d_fft_local;
	hipMalloc((void **) &d_fft_local, w * (h / 2 + 1) * sizeof(hipfftComplex));

	// Max 1024 (32 x 32) threads per block hence multiple blocks to operate on a frame
	dim3 blockDim(32, 32, 1);
	dim3 gridDim((int)ceil(width/32.0), (int)ceil(height/32.0), 1);

	// cuFFT plan
	hipfftHandle plan;
	if ((hipfftPlan2d(&plan, w, h, HIPFFT_R2C)) != HIPFFT_SUCCESS) {
		std::cout << "cuFFT Plan Error" << std::endl;
	}

	// Main loop

	int tau, idx1, idx2;
	float *d_frame1, *d_frame2;

	for (int repeats = 0; repeats < 10; repeats++) {
		for (int tau_idx = 0; tau_idx < tau_count; tau_idx++) {
			tau = tau_vector[tau_idx];

			idx1 = rand() % (frame_count - tau);
			idx2 = idx1 + tau;
			std::cout << "tau: " << tau << " idxs: " << idx1 << ", " << idx2 << std::endl;

			d_frame1 = d_chunk_ptr + (idx1 * w * h);	// float pointer to frame 1
			d_frame2 = d_chunk_ptr + (idx2 * w * h);

			AbsDifference<<<gridDim, blockDim>>>(d_diff_local, d_frame1, d_frame2, w, h); // find absolute difference

			// FFT execute
			if ((hipfftExecR2C(plan, d_diff_local, d_fft_local)) != HIPFFT_SUCCESS) {
				std::cout << "cuFFT Exec Error" << std::endl;
			}

			processFFT<<<gridDim, blockDim>>>(d_fft_local, d_out, tau_idx, w, h); // process FFT (i.e. normalise and add to accumulator)
		}
	}

//	int tau, frame1, frame2;
//
//	for (int tau_idx = 0; tau_idx < tau_count; tau_idx++) {
//		for (int repeats = 0; repeats < 40; repeats++) {
//			tau = tau_vector[tau_idx];
//
//			frame1 = rand() % (chunk_frame_count - tau);
//			frame2 = frame1 + tau;
//
//
//
//			std::cout << " Abs Diff" << std::endl;
//			AbsDifference<<<gridDim, blockDim>>>(d_buffer, d_diff_local, frame1, frame2, width, height);
//
//			// FFT
//			std::cout << " FFt Diff" << std::endl;
//			hipfftHandle plan;
//			if ((hipfftPlan2d(&plan, height, width, HIPFFT_R2C)) != HIPFFT_SUCCESS) {
//				std::cout << "cufft plan error" << std::endl;
//			}
//
//			if ((hipfftExecR2C(plan, (hipfftReal*)d_diff_local, (hipfftComplex*)d_fft_local)) != HIPFFT_SUCCESS) {
//				std::cout << "cufft exec error" << std::endl;
//			}
//
//			std::cout << " Process FFt" << std::endl;
//			processFFT<<<gridDim, blockDim>>>(d_fft_local, d_fft_accum, tau_idx, width, height);
//			//hipMemcpy(print_buffer, d_fft_accum, width*height*sizeof(float), hipMemcpyDeviceToHost);
//		}
//	}

}

void RunDDM(float *out, VideoCapture cap, int width, int height, int frame_count, int tau_count, int* tau_vector, float * print_buffer) {
	std::cout<<"Start"<<std::endl;

	// Initialise buffer parameters
	int buff_frames, chunk_frames;
	buff_frames = 30; // as we are running load / analyse in serial buff size = chunk size
	chunk_frames = 30;

	int num_data = width * height;
	int buff_size = num_data * buff_frames * sizeof(float);
	int work_size = num_data * tau_count * sizeof(float);

	float *d_buffer, *d_fftAccum;

	// allocate device memory for buffer and output
	hipMalloc((void **) &d_buffer, buff_size);
	hipMalloc((void **) &d_fftAccum, work_size);

	// At the moment we run each operation in series - can parallelise later

	while (frame_count >= chunk_frames) {
		std::cout << frame_count << " Frames left" << std::endl;
		LoadVideoToBuffer(d_buffer, chunk_frames, cap, width, height);
		analyseChunk(d_buffer, chunk_frames, d_fftAccum, tau_vector, tau_count, width, height, print_buffer);
		frame_count -= chunk_frames;
	}

	hipMemcpy(out, d_fftAccum, work_size, hipMemcpyDeviceToHost);
	std::cout<<"Done"<<std::endl;
}

void HARDCODEanalyseFFTHost(float *d_in, float *d_out, int *tau_vector, int tau_count, int width, int height, float *debug_buff=NULL) {
    int w = width; int h = height;

	// Generate q - vectors - Hard Coded
	int q_count = 20;
	float q_squared[20] = 	{ 1.        ,   3.6472384 ,   7.94985584,  13.90785234,  21.52122788, 30.78998247,  41.71411612,
    							 54.29362881,  68.52852055,  84.41879134, 101.96444118, 121.16547007, 142.021878 , 164.53366499,
								188.70083102, 214.52337611, 242.00130024, 271.13460343, 301.92328566, 334.36734694};
	// Generate masks
    int *px_count = new int[q_count](); // () initialises to zero
    float *masks = new float[w * h * q_count];

    float half_w, half_h;
    half_h = height / 2.0;
    half_w = width / 2.0;
    float r_sqr, ratio;

    // First Generate the radius masks
    int shift_x, shift_y;
    for (int q_idx = 0; q_idx < q_count; q_idx++) {
        for (int x = 0; x < w; x++)
        {
            for (int y = 0; y < h; y++)
            {
                // Perform manual FFT shift
                shift_x = (x + (int)half_w) % w;
                shift_y = (y + (int)half_h) % h;

                // Distance relative to centre
                shift_x -= half_w;
                shift_y -= half_h;

                r_sqr = shift_x * shift_x + shift_y * shift_y;
                ratio = r_sqr / q_squared[q_idx];

                if (1 <= ratio && ratio <= 1.44) { // we want values from 1.0 * q to 1.2 * q
                    masks[q_idx*w*h + y*w + x] = 1.0;
                    px_count[q_idx] += 1;
                } else {
                    masks[q_idx*w*h + y*w + x] = 0.0;
                }
            }
        }
    }
//    // Mask generation end
//
//    // Start analysis
//    float *tau_frame;
//    float val;
//
//    for (int tau_idx = 0; tau_idx < tau_count; tau_idx++) {
//        tau_frame = d_in + (w * h * tau_idx);
//
//        for (int q_idx = 0; q_idx < q_count; q_idx++) {
//        	val = 0;
//        	if (px_count[q_idx] != 0) { // If the mask has no values iq_tau must be zero
//                for (int i = 0; i < w*h; i++) { 	// iterate through all pixels
//                	val += d_in[w * h * tau_idx + i] * masks[w * h * tau_idx + i];
//                }
//                // Also should divide by chunk count
//                val /= (float)px_count[q_idx]); // could be potential for overflow here
//        	}
//
//        	iq_tau[q_idx * tau_count + tau_idx] = val;
//        }
//    }
//
//


}






void analyseFFTHost(float *h_in, float *iq_tau, int number_chunks, int tau_count, int* tau_vector, int width, int height) {
	// Handles generation of masks
	std::cout << "Final analysis start" << std::endl;

	int frame_size = width * height;


//	int smallest_size = (width < height) ? width : height;
//	int q_count = (int)(log2(smallest_size) + log2(2.0/3.0) - 1);
//    float *q_vector = new float[q_count];
//    float *q_sq_vector = new float[q_count];
//    int current_q = 3;
//    for (int i=0; i < q_count; i++) {
//        std::cout << current_q << std::endl;
//        q_vector[i] =  current_q;
//        q_sq_vector[i] = q_vector[i] * q_vector[i];
//        current_q *= 2;
//    }

	int q_count = 20;
    float q_vector[20] = { 1.        ,  1.90977444,  2.81954887,  3.72932331,  4.63909774,
							  5.54887218,  6.45864662,  7.36842105,  8.27819549,  9.18796992,
							 10.09774436, 11.0075188 , 11.91729323, 12.82706767, 13.73684211,
							 14.64661654, 15.55639098, 16.46616541, 17.37593985, 18.28571429};

    float q_sq_vector[20] = {  1.        ,   3.6472384 ,   7.94985584,  13.90785234,  21.52122788,
								  30.78998247,  41.71411612,  54.29362881,  68.52852055,  84.41879134,
								 101.96444118, 121.16547007, 142.021878  , 164.53366499, 188.70083102,
								 214.52337611, 242.00130024, 271.13460343, 301.92328566, 334.36734694};

    int *px_count = new int[q_count]();
    float *masks = new float[frame_size * q_count];

    float half_w, half_h;
    half_h = height / 2.0;
    half_w = width / 2.0;
    float r_sqr, ratio;

    // First Generate the radius masks
    int shift_x, shift_y;
    for (int q_idx = 0; q_idx < q_count; q_idx++) {
        for (int x = 0; x < width; x++)
        {
            for (int y = 0; y < height; y++)
            {
                // We want the x and y values to be FFT shifted, we can perform this manually
                shift_x = (x + (int)half_w) % width;
                shift_y = (y + (int)half_h) % height;

                r_sqr = (shift_x - half_w) * (shift_x - half_w) + (shift_y - half_h) * (shift_y - half_h);
                ratio = r_sqr / q_sq_vector[q_idx];
                if (1 <= ratio && ratio <= 1.44) {
                    masks[q_idx*frame_size + y*width + x] = 1.0;
                    px_count[q_idx] += 1;
                } else {
                    masks[q_idx*frame_size + y*width + x] = 0.0;
                }
            }
        }
    }
    // Mask generation end

    // Start analysis
    float * tau_frame;
    float val;

    for (int tau_idx = 0; tau_idx < tau_count; tau_idx++) {
        tau_frame = h_in + (frame_size * tau_idx);

        for (int q_idx = 0; q_idx < q_count; q_idx++) {
        	val = 0;
        	if (!(px_count[q_idx] == 0)) { // If the mask has no values iq_tau must be zero
                for (int i = 0; i < frame_size; i++) { 	// iterate through all pixels
                	val += tau_frame[i] * masks[q_idx * frame_size + i];
                }
                val /= ((float)number_chunks * (float)px_count[q_idx]); // could be potential for overflow here
        	}

        	iq_tau[q_idx * tau_count + tau_idx] = val;
        }
    }

}

int main(int argc, char **argv)
{
	VideoCapture cap("/home/ghaskell/projects_Git/cuDDM/data/colloid_0.5um_vid.mp4");

	int tau_count = 11;
	int tau_vector [tau_count] = {2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12};
	int width = 512;
	int height = 512;
	int frame_count = 400;


	float * out = new float [width * height * tau_count];
	float * print_buffer = new float[width * height];
	RunDDM(out, cap, width, height, frame_count, tau_count, &tau_vector[0], print_buffer);
	write_to_file(out, width, height);


	// HARD CODED - BAD - only works for 1024
	int q_count = 20;
    float q_vector[20] = { 1.        ,  1.90977444,  2.81954887,  3.72932331,  4.63909774,
							  5.54887218,  6.45864662,  7.36842105,  8.27819549,  9.18796992,
							 10.09774436, 11.0075188 , 11.91729323, 12.82706767, 13.73684211,
							 14.64661654, 15.55639098, 16.46616541, 17.37593985, 18.28571429};

	float * iq_tau = new float[tau_count * q_count]();

	analyseFFTHost(out, iq_tau, 20, tau_count, tau_vector, width, height);

	for (int i = 0; i < tau_count * q_count; i++) {
		std::cout << iq_tau[i] << std::endl;
	}

	// outputting iqtau
    std::ofstream myfile("/home/ghaskell/projects_Git/cuDDM/data/iqt.txt");

    if (myfile.is_open()) {
    	for (int i = 0; i < q_count; i++) {
    		myfile << q_vector[i] << " ";
    	}
		myfile << "\n";
    	for (int i = 0; i < tau_count; i++) {
    		myfile << tau_vector[i] << " ";
    	}
		myfile << "\n";

		for (int q_idx = 0; q_idx < q_count; q_idx++) {
	    	for (int t_idx = 0; t_idx < tau_count; t_idx++) {
	    		myfile << out[q_idx * tau_count + t_idx] << " ";
	    	}
			myfile << "\n";
		}

		myfile.close();
    } else {
    	std::cout << "Unable to open file";
    	return 0;
    }


//	std::ofstream myfile("/home/ghaskell/projects_Git/cuDDM/data/data2.txt");
//	float * print_buff = new float[width * height];
//	if (myfile.is_open()) {
//		for (int t = 0; t < 1; tau_count++) {
//			for (int x = 0; x < width*height; x++) {
//				myfile << out[t*width*height+ x] <<" ";
//			}
//			myfile << std::endl;
//		}
//	}
//	myfile.close();

	std::cout << "DONE" << std::endl;

}


